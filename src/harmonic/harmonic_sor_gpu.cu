#include "hip/hip_runtime.h"
/**
 *  The MIT License (MIT)
 *
 *  Copyright (c) 2014 Kyle Hollins Wray, University of Massachusetts
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy of
 *  this software and associated documentation files (the "Software"), to deal in
 *  the Software without restriction, including without limitation the rights to
 *  use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 *  the Software, and to permit persons to whom the Software is furnished to do so,
 *  subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in all
 *  copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 *  FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 *  COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 *  IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 *  CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */


#include "harmonic.h"
#include "harmonic_sor_gpu.h"
#include "error_codes.h"
#include "constants.h"


void harmonic_sor_2d_update_gpu(unsigned int *m, float *u)
{

}

int harmonic_sor_2d_gpu(Harmonic *harmonic, unsigned int numThreads)
{
    // Ensure data is valid before we begin.
    if (harmonic == nullptr || harmonic->m == nullptr || harmonic->u == nullptr ||
            harmonic->locked == nullptr || harmonic->epsilon <= 0.0 ||
            harmonic->omega < 1.0 || harmonic->omega >= 2.0) {
        fprintf(stderr, "Error[harmonic_sor_2d_cpu]: %s\n", "Invalid data.");
        return INERTIA_ERROR_INVALID_DATA;
    }

    // Make sure 'information' can at least be propagated throughout the entire grid.
    unsigned int mMax = 0;
    for (unsigned int i = 0; i < harmonic->n; i++) {
        mMax = std::max(mMax, harmonic->m[i]);
    }

    harmonic->currentIteration = 0;

    float delta = harmonic->epsilon + 1.0;
    while (delta > harmonic->epsilon || harmonic->currentIteration < mMax) {
        delta = 0.0;

        // Iterate over all non-boundary cells and update its value based on a red-black ordering.
        // Thus, for all rows, we either skip by evens or odds in 2-dimensions.
        for (unsigned int x0 = 1; x0 < harmonic->m[0] - 1; x0++) {
            // Determine if this rows starts with a red (even row) or black (odd row) cell, and
            // update the opposite depending on how many iterations there have been.
            bool redRow = (x0 % 2 == 0);
            bool evenIteration = (harmonic->currentIteration % 2 == 0);
            unsigned int offset = 0; //(unsigned int)((harmonic->currentIteration % 2) != (x0 % 2));

            if ((evenIteration && !redRow) || (!evenIteration && redRow)) {
                offset = 1;
            }

            for (unsigned int x1 = 1 + offset; x1 < harmonic->m[1] - 1; x1 += 2) {
                // If this is locked, then skip it.
                if (harmonic->locked[x0 * harmonic->m[1] + x1]) {
                    continue;
                }

                float uPrevious = harmonic->u[x0 * harmonic->m[1] + x1];

                // Update the value at this location with the log-sum-exp trick.
                float maxVal = FLT_MIN;
                maxVal = std::max(harmonic->u[(x0 - 1) * harmonic->m[1] + x1], harmonic->u[(x0 + 1) * harmonic->m[1] + x1]);
                maxVal = std::max(maxVal, harmonic->u[x0 * harmonic->m[1] + (x1 - 1)]);
                maxVal = std::max(maxVal, harmonic->u[x0 * harmonic->m[1] + (x1 + 1)]);

                harmonic->u[x0 * harmonic->m[1] + x1] =  maxVal + std::log(
                                                            std::exp(harmonic->u[(x0 - 1) * harmonic->m[1] + x1] - maxVal) +
                                                            std::exp(harmonic->u[(x0 + 1) * harmonic->m[1] + x1] - maxVal) +
                                                            std::exp(harmonic->u[x0 * harmonic->m[1] + (x1 - 1)] - maxVal) +
                                                            std::exp(harmonic->u[x0 * harmonic->m[1] + (x1 + 1)] - maxVal)) -
                                                        std::log(2.0 * harmonic->n);

                // Compute the updated delta.
                delta = std::max(delta, (float)fabs(uPrevious - harmonic->u[x0 * harmonic->m[1] + x1]));
            }
        }

        // *** DEBUG ***
        if (harmonic->currentIteration % 100 == 0) {
            printf("Iteration %i --- %e\n", harmonic->currentIteration, delta);
            fflush(stdout);
        }
        // *************

        harmonic->currentIteration++;
    }

    return INERTIA_SUCCESS;
}

//int harmonic_sor_3d_gpu(Harmonic *harmonic, unsigned int numThreads);

//int harmonic_sor_4d_gpu(Harmonic *harmonic, unsigned int numThreads);



