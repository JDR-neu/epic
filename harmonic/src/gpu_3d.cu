#include "hip/hip_runtime.h"
/**
 *  The MIT License (MIT)
 *
 *  Copyright (c) 2014 Kyle Hollins Wray, University of Massachusetts
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy of
 *  this software and associated documentation files (the "Software"), to deal in
 *  the Software without restriction, including without limitation the rights to
 *  use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 *  the Software, and to permit persons to whom the Software is furnished to do so,
 *  subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in all
 *  copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 *  FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 *  COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 *  IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 *  CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */


#include <iostream>

#include "../include/gpu.h"

__global__ void gpu_harmonic_check_3d(unsigned int *m, float *u, float *uPrime, float epsilon, unsigned int *running)
{
	for (unsigned int i = blockIdx.x; i < m[0]; i += gridDim.x) {
		for (unsigned int j = threadIdx.x; j < m[1]; j += blockDim.x) {
			for (unsigned int k = threadIdx.y; k < m[2]; k += blockDim.y) {
				// We need to keep looping if this is not an obstacle, and the difference
				// between iterations was greater than the tolerance.
				if (signbit(u[i * m[1] * m[2] + j * m[2] + k]) == 0 &&
						fabsf(uPrime[i * m[1] * m[2] + j * m[2] + k] -
								u[i * m[1] * m[2] + j * m[2] + k]) > epsilon) {
					*running = 1;
				}
			}
		}
	}
}

__global__ void gpu_harmonic_iteration_3d(unsigned int *m, float *u, float *uPrime, float epsilon)
{
	for (unsigned int i = blockIdx.x; i < m[0]; i += gridDim.x) {
		for (unsigned int j = threadIdx.x; j < m[1]; j += blockDim.x) {
			for (unsigned int k = threadIdx.y; k < m[2]; k += blockDim.y) {
				// Skip this if it is an obstacle. Perhaps it is better to actually just wastefully compute the
				// equations below, instead of causing branch divergence.
				if (signbit(u[i * m[1] * m[2] + j * m[2] + k]) == 0) {
					// Since this solver assumes the boundary is fixed, we do not need to check min and max.
					// Unless, you decide to merge the if statement into the equations below... then you need these.
//					unsigned int ip = min(m[0] - 1, i + 1);
//					unsigned int im = max(0, (int)i - 1);
//					unsigned int jp = min(m[1] - 1, j + 1);
//					unsigned int jm = max(0, (int)j - 1);
//					unsigned int kp = min(m[2] - 1, k + 1);
//					unsigned int km = max(0, (int)k - 1);

					float val = 0.16666666667f *
							(fabsf(u[(i + 1) * m[1] * m[2] + j * m[2] + k]) +
							fabsf(u[(i - 1) * m[1] * m[2] + j * m[2] + k]) +
							fabsf(u[i * m[1] * m[2] + (j + 1) * m[2] + k]) +
							fabsf(u[i * m[1] * m[2] + (j - 1) * m[2] + k]) +
							fabsf(u[i * m[1] * m[2] + j * m[2] + (k + 1)]) +
							fabsf(u[i * m[1] * m[2] + j * m[2] + (k - 1)]));

					// TODO: Convert this into a separate kernel with the first element assigning the boolean running to false.
					// Then sync threads. Then set running to true if fabs(u[] - uPrime[]) > epsilon. Make running an unsigned int...
//					*running = *running + (unsigned long long int)(fabsf(val - u[i * m[1] * m[2] + j * m[2] + k]) > epsilon);

					uPrime[i * m[1] * m[2] + j * m[2] + k] = val;
				}
			}
		}
	}
}

int gpu_harmonic_alloc_3d(unsigned int *m, float *u,
		unsigned int *&d_m, float *&d_u, float *&d_uPrime)
{
	// Ensure the data is valid.
	if (u == nullptr || m == nullptr || m[0] == 0 || m[1] == 0 || m[2] == 0) {
		std::cerr << "Error[gpu_harmonic_alloc_3d]: Invalid data." << std::endl;
		return 1;
	}

	// Allocate the memory on the device.
	if (hipMalloc(&d_m, 3 * sizeof(unsigned int)) != hipSuccess) {
		std::cerr << "Error[gpu_harmonic_alloc_3d]: Failed to allocate device-side memory for the dimension size values." << std::endl;
		return 2;
	}
	if (hipMalloc(&d_u, m[0] * m[1] * m[2] * sizeof(float)) != hipSuccess) {
		std::cerr << "Error[gpu_harmonic_alloc_3d]: Failed to allocate device-side memory for the harmonic function values." << std::endl;
		return 2;
	}
	if (hipMalloc(&d_uPrime, m[0] * m[1] * m[2] * sizeof(float)) != hipSuccess) {
		std::cerr << "Error[gpu_harmonic_alloc_3d]: Failed to allocate device-side memory for the harmonic function values." << std::endl;
		return 2;
	}

	// Copy the data from the host to the device. Note: Even if things like d_uPrime get overwritten,
	// you MUST malloc AND memcpy to use them!
	if (hipMemcpy(d_m, m, 3 * sizeof(unsigned int), hipMemcpyHostToDevice) != hipSuccess) {
		std::cerr << "Error[gpu_harmonic_alloc_3d]: Failed to copy memory from host to device for the dimension size function." << std::endl;
		return 3;
	}
	if (hipMemcpy(d_u, u, m[0] * m[1] * m[2] * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		std::cerr << "Error[gpu_harmonic_alloc_3d]: Failed to copy memory from host to device for the harmonic function." << std::endl;;
		return 3;
	}
	if (hipMemcpy(d_uPrime, u, m[0] * m[1] * m[2] * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		std::cerr << "Error[gpu_harmonic_alloc_3d]: Failed to copy memory from host to device for the harmonic function (prime)." << std::endl;
		return 3;
	}

	return 0;
}

int gpu_harmonic_execute_3d(unsigned int *m, float epsilon,
		unsigned int *d_m, float *d_u, float *d_uPrime,
		unsigned int numBlocks, unsigned int numThreadsX, unsigned int numThreadsY,
		unsigned int stagger)
{
	// Ensure the data is valid.
	if (m == nullptr || epsilon <= 0.0f || d_m == nullptr || d_u == nullptr || numThreadsX == 0 || numThreadsY == 0) {
		std::cerr << "Error[gpu_harmonic_execute_3d]: Invalid data." << std::endl;
		return 1;
	}

	// Also ensure that the number of threads executed are valid.
	if (numThreadsX * numThreadsY % 32 != 0) {
		std::cerr << "Error[gpu_harmonic_execute_3d]: Must specify a number of threads divisible by 32 (the number of threads in a warp)." << std::endl;
		return 1;
	}

	// We must ensure that the stagger for convergence checking is even (i.e., num iterations), so that d_u stores the final result, not d_uPrime.
	if (stagger % 2 == 1) {
		std::cerr << "Error[gpu_harmonic_execute_3d]: Stagger for convergence checking must be even." << std::endl;
		return 1;
	}

	// Create the running value, which keeps the iterations going so long as at least one element needs updating.
	unsigned int *running = new unsigned int;
	*running = 1;

	unsigned int *d_running = nullptr;
	if (hipMalloc(&d_running, sizeof(unsigned int)) != hipSuccess) {
		std::cerr << "Error[gpu_harmonic_execute_3d]: Failed to allocate device-side memory for the running variable." << std::endl;
		return 2;
	}

	if (hipMemcpy(d_running, running, sizeof(unsigned int), hipMemcpyHostToDevice) != hipSuccess) {
		std::cerr << "Error[gpu_harmonic_execute_3d]: Failed to copy running object from host to device." << std::endl;
		return 3;
	}

	// Iterate until convergence.
	unsigned long long int iterations = 0;

	// Important Note: Must ensure that iterations is even so that d_u stores the final result, not d_uPrime.
	while (*running > 0) {
		// Perform one step of the iteration, either using u and storing in uPrime, or vice versa.
		if (iterations % 2 == 0) {
			gpu_harmonic_iteration_3d<<< numBlocks, dim3(numThreadsX, numThreadsY) >>>(d_m, d_u, d_uPrime, epsilon);
		} else {
			gpu_harmonic_iteration_3d<<< numBlocks, dim3(numThreadsX, numThreadsY) >>>(d_m, d_uPrime, d_u, epsilon);
		}
		if (hipGetLastError() != hipSuccess) {
			std::cerr << "Error[gpu_harmonic_execute_3d]: Failed to execute the 'iteration' kernel." << std::endl;
			return 3;
		}

		// Wait for the kernel to finish before looping more.
		if (hipDeviceSynchronize() != hipSuccess) {
			std::cerr << "Error[gpu_harmonic_execute_3d]: Failed to synchronize the device." << std::endl;
			return 3;
		}

		// Copy the running value computed by each thread back to the host.
		if (iterations % stagger == 0) {
			*running = 0;

			if (hipMemcpy(d_running, running, sizeof(unsigned int), hipMemcpyHostToDevice) != hipSuccess) {
				std::cerr << "Error[gpu_harmonic_execute_3d]: Failed to copy running object from host to device." << std::endl;
				return 3;
			}

			gpu_harmonic_check_3d<<< numBlocks, dim3(numThreadsX, numThreadsY) >>>(d_m, d_u, d_uPrime, epsilon, d_running);
			if (hipGetLastError() != hipSuccess) {
				std::cerr << "Error[gpu_harmonic_execute_3d]: Failed to execute the 'check' kernel." << std::endl;
				return 3;
			}

			if (hipDeviceSynchronize() != hipSuccess) {
				std::cerr << "Error[gpu_harmonic_execute_3d]: Failed to synchronize the device." << std::endl;
				return 3;
			}

			if (hipMemcpy(running, d_running, sizeof(unsigned int), hipMemcpyDeviceToHost) != hipSuccess) {
				std::cerr << "Error[gpu_harmonic_execute_3d]: Failed to copy running object from device to host." << std::endl;
				return 3;
			}
		}

		iterations++;
	}

//	std::cout << "Completed in " << iterations << " iterations." << std::endl;

	// Free the memory of the delta value.
	delete running;
	if (hipFree(d_running) != hipSuccess) {
		std::cerr << "Error[gpu_harmonic_execute_3d]: Failed to free memory for the running flag." << std::endl;
		return 4;
	}

	return 0;
}

int gpu_harmonic_get_3d(unsigned int *m, float *d_u, float *u)
{
	if (hipMemcpy(u, d_u, m[0] * m[1] * m[2] * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
		std::cerr << "Error[gpu_harmonic_get_3d]: Failed to copy memory from device to host for the entire result." << std::endl;
		return 1;
	}
	return 0;
}

int gpu_harmonic_free_3d(unsigned int *d_m, float *d_u, float *d_uPrime)
{
	if (hipFree(d_m) != hipSuccess) {
		std::cerr << "Error[gpu_harmonic_free_3d]: Failed to free memory for the dimension sizes." << std::endl;
		return 1;
	}
	if (hipFree(d_u) != hipSuccess) {
		std::cerr << "Error[gpu_harmonic_free_3d]: Failed to free memory for the harmonic function." << std::endl;
		return 1;
	}
	if (hipFree(d_uPrime) != hipSuccess) {
		std::cerr << "Error[gpu_harmonic_free_3d]: Failed to free memory for the harmonic function (prime)." << std::endl;
		return 1;
	}
	return 0;
}
